#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <string.h>
#include <math.h>
#include <float.h>
#include <time.h>

// includes, kernels
#include "vector_dot_product_kernel.cu"

void run_test(unsigned int);
void compute_on_device(float *, float *,float *,int);
extern "C" float compute_gold( float *, float *, unsigned int);

int 
main( int argc, char** argv) 
{
	if(argc != 2){
		printf("Usage: vector_dot_product <num elements> \n");
		exit(0);	
	}
	unsigned int num_elements = atoi(argv[1]);
	run_test(num_elements);
	return 0;
}

void 
run_test(unsigned int num_elements) 
{
	// Obtain the vector length
	unsigned int size = sizeof(float) * num_elements;

	// Allocate memory on the CPU for the input vectors A and B
	float *A = (float *)malloc(size);
	float *B = (float *)malloc(size);
	float *C = (float *)malloc(NUM_BLOCKS);
	float gpu_result = 0.0f;
	// Randomly generate input data. Initialize the input data to be floating point values between [-.5 , 5]
	printf("Generating random vectors with values between [-.5, .5]. \n");	
	srand(time(NULL));
	for(unsigned int i = 0; i < num_elements; i++){
		A[i] = (float)rand()/(float)RAND_MAX - 0.5;
		B[i] = (float)rand()/(float)RAND_MAX - 0.5;
	}
	for(unsigned int i = 0; i < NUM_BLOCKS; i++){
		C[i] = 0.0f;
	}
	printf("Generating dot product on the CPU. \n");
	float reference = compute_gold(A, B, num_elements);
    
	/* Edit this function to compute the result vector on the GPU. 
       The result should be placed in the gpu_result variable. */
	compute_on_device(A, B, C, num_elements);
	for(unsigned int i = 0; i<NUM_BLOCKS; i++){
		gpu_result += C[i];
	}

	printf("Result on CPU: %f, result on GPU: %f. \n", reference, gpu_result);
    printf("Epsilon: %f. \n", fabsf(reference - gpu_result));

	// cleanup memory
	free(A);
	free(B);
	free(C);
	return;
}

/* Edit this function to compute the dot product on the device using atomic intrinsics. */
void 
compute_on_device(float *A_on_host, float *B_on_host, float *C_on_host, int num_elements)
{
	float *A_on_device = NULL;
	float *B_on_device = NULL;
	float *C_on_device = NULL; 
	
	hipMalloc((void**)&A_on_device, num_elements * sizeof(float));
	hipMemcpy(A_on_device, A_on_host, num_elements * sizeof(float), hipMemcpyHostToDevice);

	hipMalloc((void**)&B_on_device, num_elements * sizeof(float));
	hipMemcpy(B_on_device, B_on_host, num_elements * sizeof(float), hipMemcpyHostToDevice);

	// Allocate space for the result vector on the GPU
	hipMalloc((void**)&C_on_device, NUM_BLOCKS * sizeof(float));
	hipMemcpy(C_on_device, C_on_host, NUM_BLOCKS * sizeof(float), hipMemcpyHostToDevice);
	// Set up the execution grid on the GPU
	dim3 thread_block(THREAD_BLOCK_SIZE, 1, 1); // Set the number of threads in the thread block
	dim3 grid(NUM_BLOCKS,1);
	
	// Launch the kernel
	vector_dot_product_kernel<<<grid, thread_block>>>(A_on_device, B_on_device, C_on_device, num_elements);

	hipMemcpy(C_on_host, C_on_device, NUM_BLOCKS * sizeof(float), hipMemcpyDeviceToHost);

	// Free memory
	hipFree(A_on_device);
	hipFree(B_on_device);
	hipFree(C_on_device);
}
 

