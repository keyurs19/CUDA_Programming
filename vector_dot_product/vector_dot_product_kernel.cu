
#include <hip/hip_runtime.h>

#define THREAD_BLOCK_SIZE 1024
#define NUM_BLOCKS 160 // Define the size of a tile
__global__ void vector_dot_product_kernel(float *A, float *B, float *C, unsigned int num_elements)
{
	__shared__ float sum_per_thread[THREAD_BLOCK_SIZE];	
	unsigned int thread_id = blockIdx.x * blockDim.x + threadIdx.x; // Obtain the index of the thread
	unsigned int stride = blockDim.x * gridDim.x; 
	float sum = 0.0f; 
	unsigned int i = thread_id; 

	while(i < num_elements){
			  sum += A[i] * B[i];
			  i += stride;
	}

	sum_per_thread[threadIdx.x] = sum; // Copy sum to shared memory
	__syncthreads();

	i = blockDim.x/2;
	while(i != 0){
			  if(threadIdx.x < i) 
						 sum_per_thread[threadIdx.x] += sum_per_thread[threadIdx.x + i];
			  __syncthreads();
			  i /= 2;
	}

	if(threadIdx.x == 0)
			  C[blockIdx.x] = sum_per_thread[0];
}
